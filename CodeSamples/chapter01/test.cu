#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>

 __global__ void incr(int *ptr)
 {
   int tmp=*ptr;
   tmp=tmp+1;
   *ptr=tmp;
   printf("%d",*ptr);
 }

 __global__ void helloFromGPU()
 {
    printf("Hello World from GPU!\n");
 }
 
 int main(int argc, char **argv)
 {
    printf("Hello World from CPU!\n");
    int a = 12413;
    int *p;
    p=&a;
    helloFromGPU<<<1, 3>>>();
    incr<<<1, 1>>>(p);
    CHECK(hipDeviceReset());
    return 0;
 }
 
