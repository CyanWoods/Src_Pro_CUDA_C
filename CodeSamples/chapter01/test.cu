#include "hip/hip_runtime.h"

#include <stdio.h>

hipMalloc(*temp,sizeof(int)*3);

 __global__ void incr(int *ptr)
 {
   printf("!\n");
   temp[threadIdx.x]=atomicAdd(ptr,1);
   printf("%d\n",temp[threadIdx.x]);
 }

 __global__ void helloFromGPU()
 {
    printf("Hello World from GPU!\n");
 }
 
 int main(int argc, char **argv)
 {
    
    printf("Hello World from CPU!\n");
    int a = 12413;
    int *p;
    p=&a;
    helloFromGPU<<<1, 3>>>();
    incr<<<1, 3>>>(p);
    hipDeviceReset();
    return 0;
 }
 
